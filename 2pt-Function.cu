/*
 ============================================================================
 Name        : 2pt-Function.cu
 Author      : Michael Cammarere & Matt Bellis
 Version     :
 Copyright   : 
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <math.h>
#include<stdio.h>

#define imax 16
#define FLOAT_RAND_MAX (float)RAND_MAX

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0/data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

float *cpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
	return rc;
}


void initialize(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5*(i+1);
}

/*This function finds the minimum of an array of floats.
 * It takes the input of the array.
 */

float* ArrMin(float *arr, int size)
{
	//Declaring min & setting it as the 1st value in the array
	float *minVal = &arr[0];

	//For loop to find the minimum
	for (int count = 1; count < size; count++)
	{
		if (&arr[count] < minVal)
		{
			minVal = &arr[count];
		}
	}
	return minVal;
}

/*This function finds the maximum of an array of floats.
 * It takes the input of the array.
 */

float* ArrMax(float *arr, int size)
{
	//Declaring max & setting it as the 1st value in the array
	float* maxVal = &arr[0];

	//For loop to find the minimum
	for (int count = 1; count < size; count++)
	{
		if (&arr[count] > maxVal)
		{
			maxVal = &arr[count];
		}
	}
	return maxVal;
}

/* This function computes the distance between 2 sets of points */
__global__
void  distance (float *distances, float  *xi, float *yi, float *zi, float *xj, float *yj, float *zj)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	distances[index] = (float) sqrt( (xi[index] - xj[index]) * (xi[index] - xj[index]) +
				(yi[index] - yj[index]) * (yi[index] - yj[index]) + (zi[index] - zj[index]) * (zi[index] - zj[index]) + 1);
}


void Bins(float vals[], int valsInBin, float min, float max, int numBins, float binWidth, unsigned long binIndices[])
{


    // The user has to make sure that filled_bins has enough memory allocated
    // for numBins integers

    int bin = -1;
    float value;

    for (int count = 0; count < valsInBin; count++) {

    	value = vals[count];

        bin = -1;

        if (value < min) {
            bin = -1;
        }
        else if (value > max) {
            bin = -999;
        }
        else {
            bin = (int)((value - min) / binWidth);
        }

        binIndices[count] = bin;
    }

    // bin_indices is a pointer so the values in it will
    // still be accessible outside of the function.

}

int main(int argc, char *argv[])
{

	static const int WORK_SIZE = 65530;
	float *data = new float[WORK_SIZE];

	initialize (data, WORK_SIZE);

	float *recCpu = cpuReciprocal(data, WORK_SIZE);
	float *recGpu = gpuReciprocal(data, WORK_SIZE);
	float cpuSum = std::accumulate (recCpu, recCpu+WORK_SIZE, 0.0);
	float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);

	/* Verify the results */
	std::cout<<"gpuSum = "<<gpuSum<< " cpuSum = " <<cpuSum<<std::endl;

	/* Free memory */
	delete[] data;
	delete[] recCpu;
	delete[] recGpu;

	//min test
	float *testArr = new float[5];
	testArr[0] = 1.0;
	testArr[1] = 2.0;
	testArr[2] = 3.0;
	testArr[3] = 4.0;
	testArr[4] = 5.0;

	float *min = new float[1];
	float *max = new float[1];

	min = ArrMin(testArr,5);
	max = ArrMax(testArr,5);

	printf("MIN = %f\n",*min);
	printf("MAX = %f\n",*max);


	float *xi = new float[1000];
	float *yi = new float[1000];
	float *zi = new float[1000];
	float *xj = new float[1000];
	float *yj = new float[1000];
	float *zj = new float[1000];

    for (int i = 0; i < 1000; i++)
    {
    	xi[i] = (float)(rand() % 100 + 1);
    	yi[i] = (float)(rand() % 100 + 1);
    	zi[i] = (float)(rand() % 100 + 1);

    	xj[i] = (float)(rand() % 100 + 1);
    	yj[i] = (float)(rand() % 100 + 1);
    	zj[i] = (float)(rand() % 100 + 1);
    }

    hipMalloc((void**)&xi, 1000 * sizeof(float*));
    hipMalloc((void**)&yi, 1000 * sizeof(float*));
    hipMalloc((void**)&zi, 1000 * sizeof(float*));
    hipMalloc((void**)&xj, 1000 * sizeof(float*));
    hipMalloc((void**)&yj, 1000 * sizeof(float*));
    hipMalloc((void**)&zj, 1000 * sizeof(float*));

	float *distances = new float[1000];

	distance<<<1, 1000>>>(distances, xi, yi, zi, xj, yj, zj);

	for (int i = 0; i < 1000; i++)
	{
		printf("distance = %f\n",distances[i]);
	}
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

