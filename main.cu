#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>

/* 2pt Correlation Function */

/*This function finds the minimum of an array of floats.
 * It takes the input of the array and an already declared minimum varable.
 */

float* ArrMin(float *arr)
{
	//Declaring the min variable
	float *min = (float*)malloc(sizeof(float));

	// Finding size of the input array
	int size = sizeof(arr) / sizeof(arr[0]);

	//Declaring the 1st value in the array the temp min
	memcpy(min,&arr[0], sizeof(float*));

	//For loop to find the minimum
	for (int count = 1; count < size; count++)
	{
		if (arr[count] < min)
		{
			min = arr[count];
		}

	}
	return min;
}

/*This function finds the maximum of an array of floats.
 * It takes the input of the array and an already declared minimum varable.
 */

float* ArrMax(float *arr)
{
	//Declaring the max variable
	float *max;

	// Finding size of the input array
	int size = sizeof(arr) / sizeof(arr[0]);

	//Declaring the 1st value in the array the temp max
	max = arr[0];

	//For loop to find the minimum
	for (int count = 1; count < size; count++)
	{
		if (arr[count] > max)
		{
			max = arr[count];
		}

	}
	return max;
}

int main()
{
	float *testArr;
	float copyArr[5] = {1.0,2.0,3.0,4.0,5.0};

	testArr = (float*)malloc(sizeof(float) * 5);

	memcpy((void**)&testArr,copyArr,sizeof(float)*5);

	float *min, *max;

	hipMemcpy(min,ArrMin(testArr),sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(max,ArrMax(testArr),sizeof(float),hipMemcpyHostToDevice);

	printf("Max = %f",max);
	printf("Min = %f",min);

}
