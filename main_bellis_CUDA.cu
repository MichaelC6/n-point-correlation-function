
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>


// The user may want to change this 
// This defines the number of bins in the histogram
#define imax 16
#define FLOAT_RAND_MAX (float)RAND_MAX

/////////////////////////////////////////////////////////////
__global__ void binning1d(float *vals, int num_vals_to_bin, float lo, float hi, int nbins, float binwidth, int *bin_indices) {

	int bin = -1;
	float v;

	// The user has to make sure that filled_bins has enough memory allocated
	// for nbins integers

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx<num_vals_to_bin) {

		v = vals[idx];
		//v = sin(log(pow(1000,v)));
		bin = -1;
		if (v<lo) {
			bin=-1;
		}
		else if (v>hi) {
			bin = -999;
		}
		else {
			bin = (int)((v-lo)/binwidth);
		}   
		//printf("value: %f\t",v);
		//printf("bin: %d\n",bin);
		bin = idx; // for DEBUGGING
		bin_indices[idx] = bin;
		//bin_indices[i] = 5;
	}

	bin_indices[4] = 5;

	// bin_indices is a pointer so the values in it will 
	// still be accessible outside of the function.

}

///////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {

	// How many random values do we want to process? 
	// This can be set on the command line or you can just
	// edit it and recompile each time
	unsigned long nvals = atoll(argv[1]);
	printf("nvals: %lu\n",nvals);


	// For the random numbers, they will be between 0 and 1
	float lo = 0;
	float hi = 1;
	int nbins = imax;
	float binwidth = (hi-lo)/nbins;
	// This means we'll send this many values to the function
	// to be histogrammed
	int histogram_chunks = 16*16; 

	printf("Filling a histogram with\n");
	printf("Range: %f-%f\n",lo,hi);
	printf("# of bins: %d\n",nbins);
	printf("Bin width: %f\n",binwidth);
	printf("We will histogram in chunks of: %d\n",histogram_chunks);

	// These will be the values we're histogramming
	// I think this is wrong and tries to allocate too much memory
	//float *values_to_be_histogrammed = (float*) malloc(nvals * sizeof(float));
	//unsigned long *bin_indices = (unsigned long*) malloc(nvals * sizeof(unsigned long));
	float *values_to_be_histogrammed = (float*) malloc(histogram_chunks * sizeof(float));
	int *bin_indices = (int*) malloc(histogram_chunks * sizeof(int));

	float *d_values_to_be_histogrammed;
	int *d_bin_indices;
	hipMalloc ((void**)&d_values_to_be_histogrammed, histogram_chunks * sizeof(float));
	hipMalloc ((void**)&d_bin_indices, histogram_chunks * sizeof(int));

	hipMemset (d_bin_indices, -1, histogram_chunks * sizeof(int));

	int i = 0; 

	static unsigned long hist[imax];

	printf("Allocated the memory for the histogram.\n");

	// Zero out the entries in the histogram
	for (i=0;i<imax;i++) {
		hist[i] = 0;
	}

	printf("Zeroed the memory in the histogram.\n");

	printf("Filling the memory with %lu entries .\n",nvals);
	printf("RAND_MAX: %d\n",RAND_MAX);

	int count_for_histogramming = 0;

	for(unsigned long count=0;count<nvals;count++)
	{
		if (count%1000000==0){
			printf("count: %lu\n",count);
		}

		// Fill the array of values that we will histogram
		values_to_be_histogrammed[count_for_histogramming] = rand()/FLOAT_RAND_MAX;

		// DEBUG PRINT
		printf("This point: %lu %f\n",count,values_to_be_histogrammed[count_for_histogramming]);

		// Keep track of this by hand
		count_for_histogramming++;

		// When we have enough, go histogram them!
		if (count_for_histogramming==histogram_chunks || count==nvals-1) {
			printf("Histogramming a chunk of values!\n");

			// Copy over
			hipMemcpy(d_values_to_be_histogrammed, values_to_be_histogrammed, sizeof(float) * histogram_chunks, hipMemcpyHostToDevice);


			printf("histogram_chunks: %d\n",histogram_chunks);
			printf("count_for_histogramming: %d\n",count_for_histogramming);
			binning1d<<<16,16>>>(d_values_to_be_histogrammed, count_for_histogramming, lo, hi, nbins, binwidth, d_bin_indices);

			hipMemcpy(bin_indices, d_bin_indices, sizeof(int) * histogram_chunks, hipMemcpyDeviceToHost);

			for (int j=0;j<count_for_histogramming;j++) {
				// DEBUG PRINT
				printf("Summing: %d %d\n",j,bin_indices[j]);
				if (bin_indices[j]>=0 && bin_indices[j]<nbins)
					hist[bin_indices[j]]++;
			}
			// Reset the counter
			count_for_histogramming = 0;
		}
	}

	// Print out the histogram
	printf("Printing out the histogram entries\n");
	unsigned long total = 0;
	for (i = 0; i <  imax; i++)  {
		printf("%d %lu\n",i, hist[i]);
		total += hist[i];
	}
	printf("Total entries: %lu\n",total);

	hipFree(d_values_to_be_histogrammed);
	hipFree(d_bin_indices);
	free(values_to_be_histogrammed);
	free(bin_indices);

	return 0;
}
